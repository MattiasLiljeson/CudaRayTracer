#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "BoundingBox.cuh"
#include "Light.cuh"
#include "Mat.cuh"
#include "Options.cuh"
#include "Scene.cuh"
#include "Sphere.cuh"
#include "Tracer.cuh"
#include "Vec.cuh"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"

Tracer::Tracer(unsigned char *surface) : surface(surface) {}

Vec3f Tracer::castRay(Ray &ray, int depth) {
    if (depth > g_options.maxDepth) {
        return g_options.backgroundColor;
    }
    Vec3f hitColor = g_options.backgroundColor;
    float tnear = INF;
    SurfaceData sd;
    if (trace(ray, sd)) {
        sd.hitPoint = ray();
        sd.st = sd.hit->getStCoords(sd.triangle, sd.uv);
        sd.n = sd.hit->getNormal(sd.hitPoint, sd.triangle, sd.uv, sd.st);

        Material::Type materialType = sd.hit->getObject()->materialType;
        switch (materialType) {
            case Material::REFLECTION_AND_REFRACTION:
                return reflectionAndRefraction(ray.dir, sd, depth);
            case Material::REFLECTION:
                return reflection(ray.dir, sd, depth);
            case Material::DIFFUSE_AND_GLOSSY:
                return diffuseAndGlossy(ray.dir, sd, depth);
            default:
                return Vec3f(1.0f, 0.5f, 0.25f);
        }
    }
    return hitColor;
}

bool Tracer::trace(Ray &ray, SurfaceData &data) {
    data.hit = nullptr;
    for (int k = 0; k < g_scene.shapeCnt; ++k) {
        bool intersected = g_scene.shapes[k].intersect(ray, data);
        if (intersected) {
            data.hit = &g_scene.shapes[k];
        }
    }
    return data.hit != nullptr;
}

__device__ float clamp(const float &lo, const float &hi, const float &v) {
    return fmax(lo, fmin(hi, v));
}

// [comment]
// Compute Fresnel equation
//
// \param I is the incident view direction
//
// \param N is the normal at the intersection point
//
// \param ior is the mateural refractive index
//
// \param[out] kr is the amount of light reflected
// [/comment]
__device__ float fresnel(const Vec3f &I, const Vec3f &N, const float &ior) {
    float cosi = clamp(-1, 1, I.dot(N));
    float etai = 1, etat = ior;
    if (cosi > 0) {
        float tmp = etat;
        etat = etai;
        etai = tmp;
    }
    // Compute sini using Snell's law
    float sint = etai / etat * sqrtf(fmax(0.f, 1 - cosi * cosi));
    // Total internal reflection
    if (sint >= 1) {
        return 1;
    } else {
        float cost = sqrtf(fmax(0.f, 1 - sint * sint));
        cosi = fabsf(cosi);
        float Rs =
            ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
        float Rp =
            ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
        return (Rs * Rs + Rp * Rp) * 0.5f;
    }
    // As a consequence of the conservation of energy, transmittance is given
    // by: kt = 1 - kr;
}

// [comment]
// Compute refraction direction using Snell's law
//
// We need to handle with care the two possible situations:
//
//    - When the ray is inside the object
//
//    - When the ray is outside.
//
// If the ray is outside, you need to make cosi positive cosi = -N.I
//
// If the ray is inside, you need to invert the refractive indices and negate
// the normal N
// [/comment]
__device__ Vec3f refract(const Vec3f &I, const Vec3f &N, const float &ior) {
    float cosi = clamp(-1, 1, I.dot(N));
    float etai = 1, etat = ior;
    Vec3f n = N;
    if (cosi < 0) {
        cosi = -cosi;
    } else {
        float tmp = etai;
        etai = etat;
        etat = tmp;
        n = -N;
    }
    float eta = etai / etat;
    float k = 1 - eta * eta * (1 - cosi * cosi);
    return k < 0 ? 0 : eta * I + (eta * cosi - sqrtf(k)) * n;
}

Vec3f Tracer::reflectionAndRefraction(const Vec3f &dir, SurfaceData &data,
                                      const int depth) {
    const Material *object = data.hit->getObject();
    Vec3f hitColor = g_options.backgroundColor;
    Vec3f reflectionDirection = data.n.reflect(dir).normalized();
    Vec3f refractionDirection = refract(dir, data.n, object->ior).normalized();
    Vec3f reflectionRayOrig =
        (reflectionDirection.dot(data.n) < 0)
            ? data.hitPoint - data.n * g_options.shadowBias
            : data.hitPoint + data.n * g_options.shadowBias;
    Vec3f refractionRayOrig =
        (refractionDirection.dot(data.n) < 0)
            ? data.hitPoint - data.n * g_options.shadowBias
            : data.hitPoint + data.n * g_options.shadowBias;
    Vec3f reflectionColor =
        castRay(Ray(reflectionRayOrig, reflectionDirection), depth + 1);
    Vec3f refractionColor =
        castRay(Ray(refractionRayOrig, refractionDirection), depth + 1);
    float kr = fresnel(dir, data.n, object->ior);
    hitColor = reflectionColor * kr + refractionColor * (1 - kr);
    return hitColor;
}

Vec3f Tracer::reflection(const Vec3f &dir, SurfaceData &data, const int depth) {
    Vec3f hitColor = g_options.backgroundColor;
    float kr = fresnel(dir, data.n, data.hit->material.ior);
    Vec3f reflectionDirection = dir.reflect(data.n).normalized();
    Vec3f reflectionRayOrig =
        (reflectionDirection.dot(data.n) < 0)
            ? (data.hitPoint + data.n * g_options.shadowBias).normalized()
            : (data.hitPoint - data.n * g_options.shadowBias).normalized();
    Ray reflectionRay(reflectionRayOrig, reflectionDirection);
    hitColor = castRay(reflectionRay, depth + 1) *
               data.hit->evalDiffuseColor(data.st) * kr;

    return hitColor;
}

Vec3f Tracer::diffuseAndGlossy(const Vec3f &dir, SurfaceData &data,
                               const int depth) {
    const Material *object = data.hit->getObject();

    Vec3f hitColor = g_options.backgroundColor;
    // [comment]
    // We use the Phong illumation model in the default case. The
    // phong model is composed of a diffuse and a specular
    // reflection component.
    // [/comment]
    Vec3f lightAmt = Vec3f(0.0f, 0.0f, 0.0f);
    Vec3f specularColor = Vec3f(0.f, 0.0f, 0.0f);
    Vec3f shadowPointOrig = (dir.dot(data.n) < 0)
                                ? data.hitPoint + data.n * g_options.shadowBias
                                : data.hitPoint - data.n * g_options.shadowBias;
    // [comment]
    // Loop over all lights in the scene and sum their contribution
    // up We also apply the lambert cosine law here though we
    // haven't explained yet what this means.
    // [/comment]
    for (uint32_t i = 0; i < g_scene.lightCnt; ++i) {
        Vec3f lightDir = g_scene.lights[i].position - data.hitPoint;
        // square of the distance between hitPoint and the light
        float lightDistance = lightDir.magnitude();
        lightDir = lightDir.normalized();
        float lambert = fmaxf(0.f, lightDir.dot(data.n));
        Shape *shadowHitObject = nullptr;
        // is the point in shadow, and is the nearest occluding
        // object closer to the object than the light itself?
        Ray shadowRay(shadowPointOrig, lightDir, lightDistance);
        bool visible = !trace(shadowRay, SurfaceData());
        lightAmt += g_scene.lights[i].intensity * lambert * visible;
        Vec3f reflectionDirection = (-lightDir).reflect(data.n);
        float dotp = fmaxf(0.f, -reflectionDirection.dot(dir));

        specularColor +=
            powf(dotp, object->specularExponent) * g_scene.lights[i].intensity;
    }

    Vec3f albedo = data.hit->evalDiffuseColor(data.st);
    hitColor = lightAmt * albedo * object->Kd + specularColor * object->Ks;
    return hitColor;
}