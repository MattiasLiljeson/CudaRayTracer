#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "BoundingBox.cuh"
#include "Light.cuh"
#include "Mat.cuh"
#include "Options.cuh"
#include "Scene.cuh"
#include "Sphere.cuh"
#include "Tracer.cuh"
#include "Vec.cuh"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"

Tracer::Tracer(unsigned char *surface) : surface(surface) {}

Vec3f Tracer::castRay(Ray &ray, int depth) {
    if (depth > g_options.maxDepth) {
        return g_options.backgroundColor;
    }
    Vec3f hitColor = g_options.backgroundColor;
    float tnear = INF;
    SurfaceData sd;
    if (trace(ray, sd)) {
        sd.hitPoint = ray();
        sd.st = sd.hit->getStCoords(sd.triangle, sd.uv);
        sd.n = sd.hit->getNormal(sd.hitPoint, sd.triangle, sd.uv, sd.st);

        Material::Type materialType = sd.hit->getObject()->materialType;
        switch (materialType) {
            case Material::REFLECTION_AND_REFRACTION:
                return reflectionAndRefraction(ray.dir, sd, depth);
            case Material::REFLECTION:
                return reflection(ray.dir, sd, depth);
            case Material::DIFFUSE_AND_GLOSSY:
                return diffuseAndGlossy(ray.dir, sd, depth);
            default:
                return Vec3f(1.0f, 0.5f, 0.25f);
        }
    }
    return hitColor;
}

bool Tracer::trace(Ray &ray, SurfaceData &data) {
    data.hit = nullptr;
    for (int k = 0; k < g_scene.shapeCnt; ++k) {
        bool intersected = g_scene.shapes[k].intersect(ray, data);
        if (intersected) {
            data.hit = &g_scene.shapes[k];
        }
    }
    return data.hit != nullptr;
}

__device__ float clamp(const float &lo, const float &hi, const float &v) {
    return fmax(lo, fmin(hi, v));
}

/**
 * Fresnel computation
 * \param i incident view direction
 * \param n normal at the hit point
 * \param ior refractactive index
 * \return the amount of reflected light
 *
 *  Due to the conservation of energy, transmittance is 1 - kr;
 */
__device__ float fresnel(const Vec3f &I, const Vec3f &N, const float &ior) {
    float cosi = clamp(-1, 1, I.dot(N));
    float etai = 1, etat = ior;
    if (cosi > 0) {
        float tmp = etat;
        etat = etai;
        etai = tmp;
    }
    // Compute sini using Snell's law
    float sint = etai / etat * sqrtf(fmax(0.f, 1 - cosi * cosi));
    // Total internal reflection
    if (sint >= 1) {
        return 1;
    } else {
        float cost = sqrtf(fmax(0.f, 1 - sint * sint));
        cosi = fabsf(cosi);
        float Rs =
            ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
        float Rp =
            ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
        return (Rs * Rs + Rp * Rp) * 0.5f;
    }
}

template <typename T>
__device__ void swap(T &a, T &b) {
    T c(a);
    a = b;
    b = c;
}

/**
 * Refraction using Snell's law
 * Two cases: inside or outside object
 */
__device__ Vec3f refract(const Vec3f &I, const Vec3f &N, const float &ior) {
    float cosi = clamp(-1, 1, I.dot(N));
    float etai = 1, etat = ior;
    Vec3f n = N;
    if (cosi < 0) {  // outside
        cosi = -cosi;
    } else {  // inside
        swap(etai, etat);
        n = -N;
    }
    float eta = etai / etat;
    float k = 1 - eta * eta * (1 - cosi * cosi);
    return k < 0 ? 0 : eta * I + (eta * cosi - sqrtf(k)) * n;
}

Vec3f Tracer::reflectionAndRefraction(const Vec3f &dir, SurfaceData &data,
                                      const int depth) {
    Vec3f reflectionDirection = data.n.reflect(dir).normalized();
    Vec3f reflectionRayOrig = invBiasedOrigin(reflectionDirection, data);
    Ray reflectionRay(reflectionRayOrig, reflectionDirection);
    Vec3f reflectionColor = castRay(reflectionRay, depth + 1);

    const float ior = data.hit->getObject()->ior;
    Vec3f refractionDirection = refract(dir, data.n, ior).normalized();
    Vec3f refractionRayOrig = invBiasedOrigin(refractionDirection, data);
    Ray refractionRay(refractionRayOrig, refractionDirection);
    Vec3f refractionColor = castRay(refractionRay, depth + 1);

    float kr = fresnel(dir, data.n, ior);
    return reflectionColor * kr + refractionColor * (1 - kr);
}

Vec3f Tracer::reflection(const Vec3f &dir, SurfaceData &data, const int depth) {
    Vec3f reflectionDirection = dir.reflect(data.n).normalized();
    Vec3f reflectionRayOrig = invBiasedOrigin(reflectionDirection, data);
    Ray reflectionRay(reflectionRayOrig, reflectionDirection);
    Vec3f reflectionColor = castRay(reflectionRay, depth + 1);

    float kr = fresnel(dir, data.n, data.hit->material.ior);
    return reflectionColor * kr;
}


__device__ float blinnPhong(const Vec3f &dir, const SurfaceData &surface,
                            const Vec3f &lightDir, const float &shininess) {
    Vec3f halfDir = (lightDir + dir).normalized();
    float specAngle = fmax(halfDir.dot(surface.n), 0.0f);
    return pow(specAngle, shininess);
}

__device__ float phong(const Vec3f &dir, const SurfaceData &surface,
                       const Vec3f &lightDir, const float &shininess) {
    Vec3f reflectDir = -lightDir.reflect(surface.n);
    float specAngle = fmax(reflectDir.dot(dir), 0.0f);
    // note that the exponent is different here
    return powf(specAngle, shininess / 4.0f);
}

__device__ Vec3f shade(const Vec3f &dir, const SurfaceData &surface,
                       const Light &light, const Vec3f &diffuseColor,
                       const Vec3f &specColor, const Vec3f &lightDir,
                       const float &distance) {
     float lambertian = fmax(lightDir.dot(surface.n), 0.0f);
     float specular = 0.0f;

     if (lambertian > 0.0f) {
        specular =
            blinnPhong(dir, surface, lightDir, 1.0f /*surface.shininess*/);
    }
     Vec3f lightColor = light.intensity;
     Vec3f diffuseComponent =
        diffuseColor * lambertian * lightColor * light.lightPower / distance;
     Vec3f specComponent =
        specColor * specular * lightColor * light.lightPower / distance;
     return diffuseComponent + specComponent;
}

/**
 * Calcualte diffuse and glossy using Phong
 */
Vec3f Tracer::diffuseAndGlossy(const Vec3f &dir, SurfaceData &data,
                               const int depth) {
    const Material *object = data.hit->getObject();
    Vec3f diffuse = data.hit->evalDiffuseColor(data.st);
    Vec3f final = Vec3f(0.0f, 0.0f, 0.0f);
    Vec3f shadowPointOrig = biasedOrigin(dir, data);
    for (int i = 0; i < g_scene.lightCnt; ++i) {
        Vec3f lightDir = g_scene.lights[i].position - data.hitPoint;
        float lightDistance = lightDir.magnitude();
        lightDir = lightDir.normalized();
        float lambert = fmaxf(0.f, lightDir.dot(data.n));

        Ray shadowRay(shadowPointOrig, lightDir, lightDistance);
        bool visible = !trace(shadowRay, SurfaceData());

        if (visible) {
            final += shade(dir, data, g_scene.lights[i], diffuse, diffuse,
                           lightDir, lightDistance);
        }





        
    //// apply gamma correction (assume ambientColor, diffuseColor and
        //// specColor have been linearized, i.e. have no gamma correction in
        //// them)
        // const float screenGamma = 2.2;
        // Vec3f colorGammaCorrected;
        // colorGammaCorrected[X] = powf(colorLinear[X], 1.0f / screenGamma);
        // colorGammaCorrected[Y] = powf(colorLinear[Y], 1.0f / screenGamma);
        // colorGammaCorrected[Z] = powf(colorLinear[Z], 1.0f / screenGamma);
        //// use the gamma corrected color in the fragment
        // return colorGammaCorrected;







        // float lambert = fmaxf(0.f, lightDir.dot(data.n));

        // const Vec3f intensity = g_scene.lights[i].intensity;
        // diffuse += intensity * lambert * visible;


    }

    return final;
}

Vec3f Tracer::biasedOrigin(const Vec3f &direction, const SurfaceData &data) {
    if (direction.dot(data.n) < 0.0f) {
        return data.hitPoint + data.n * g_options.shadowBias;
    } else {
        return data.hitPoint - data.n * g_options.shadowBias;
    }
}
Vec3f Tracer::invBiasedOrigin(const Vec3f &direction, const SurfaceData &data) {
    if (direction.dot(data.n) < 0.0f) {
        return data.hitPoint - data.n * g_options.shadowBias;
    } else {
        return data.hitPoint + data.n * g_options.shadowBias;
    }
}