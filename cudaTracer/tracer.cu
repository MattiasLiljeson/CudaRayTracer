#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hiprand_normal.h>

#include "Light.cuh"
#include "Mat.cuh"
#include "Options.cuh"
#include "Sphere.cuh"
#include "Vec.cuh"
#include ""

#include <hip/hip_runtime.h>
#include "Scene.cuh"
#include "hip/hip_runtime.h"
#include "cudaUtils.h"
#include "tracer.cuh"

enum colors { RED, GREEN, BLUE, ALPHA, COLOR_CNT };

__device__ __constant__ size_t C_PITCH;

__device__ Options g_options;
__device__ Scene g_scene;

#define INF 2e10f

///////////////scratchApixel

__device__ float clamp(const float &lo, const float &hi, const float &v) {
    return fmax(lo, fmin(hi, v));
}

// [comment]
// Compute Fresnel equation
//
// \param I is the incident view direction
//
// \param N is the normal at the intersection point
//
// \param ior is the mateural refractive index
//
// \param[out] kr is the amount of light reflected
// [/comment]
__device__ void fresnel(const Vec3f &I, const Vec3f &N, const float &ior,
                        float &kr) {
    float cosi = clamp(-1, 1, I.dot(N));
    float etai = 1, etat = ior;
    if (cosi > 0) {
        float tmp = etat;
        etat = etai;
        etai = tmp;
    }
    // Compute sini using Snell's law
    float sint = etai / etat * sqrtf(fmax(0.f, 1 - cosi * cosi));
    // Total internal reflection
    if (sint >= 1) {
        kr = 1;
    } else {
        float cost = sqrtf(fmax(0.f, 1 - sint * sint));
        cosi = fabsf(cosi);
        float Rs =
            ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
        float Rp =
            ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
        kr = (Rs * Rs + Rp * Rp) / 2;
    }
    // As a consequence of the conservation of energy, transmittance is given
    // by: kt = 1 - kr;
}

// [comment]
// Compute refraction direction using Snell's law
//
// We need to handle with care the two possible situations:
//
//    - When the ray is inside the object
//
//    - When the ray is outside.
//
// If the ray is outside, you need to make cosi positive cosi = -N.I
//
// If the ray is inside, you need to invert the refractive indices and negate
// the normal N
// [/comment]
__device__ Vec3f refract(const Vec3f &I, const Vec3f &N, const float &ior) {
    float cosi = clamp(-1, 1, I.dot(N));
    float etai = 1, etat = ior;
    Vec3f n = N;
    if (cosi < 0) {
        cosi = -cosi;
    } else {
        float tmp = etai;
        etai = etat;
        etat = tmp;
        n = -N;
    }
    float eta = etai / etat;
    float k = 1 - eta * eta * (1 - cosi * cosi);
    return k < 0 ? 0 : eta * I + (eta * cosi - sqrtf(k)) * n;
}

bool Trace::trace(const Vec3f &orig, const Vec3f &dir, float &tNear,
                  uint32_t &index, Vec2f &uv, const Shape **hit) {
    *hit = nullptr;
    for (int k = 0; k < g_scene.shapeCnt; ++k) {
        float tNearK = INF;
        int indexK;
        Vec2f uvK;
        bool intersected =
            g_scene.shapes[k].intersect(orig, dir, tNearK, indexK, uvK);
        if (intersected && tNearK < tNear) {
            *hit = &g_scene.shapes[k];
            tNear = tNearK;
            index = indexK;
            uv = uvK;
        }
    }

    return (*hit != nullptr);
}

Trace::Trace(unsigned char *surface) : surface(surface) {}

Vec3f Trace::reflectionAndRefraction(const Vec3f &dir, uint32_t &index,
                                     Vec2f &uv, Vec2f &st,
                                     const Shape *hitObject,
                                     const Vec3f &hitPoint, const Vec3f &N,
                                     const int depth) {
    const Object *object = hitObject->getObject();
    Vec3f hitColor = g_options.backgroundColor;
    Vec3f reflectionDirection = N.reflect(dir).normalized();
    Vec3f refractionDirection = refract(dir, N, object->ior).normalized();
    Vec3f reflectionRayOrig = (reflectionDirection.dot(N) < 0)
                                  ? hitPoint - N * g_options.bias
                                  : hitPoint + N * g_options.bias;
    Vec3f refractionRayOrig = (refractionDirection.dot(N) < 0)
                                  ? hitPoint - N * g_options.bias
                                  : hitPoint + N * g_options.bias;
    Vec3f reflectionColor =
        castRay(reflectionRayOrig, reflectionDirection, depth + 1);
    Vec3f refractionColor =
        castRay(refractionRayOrig, refractionDirection, depth + 1);
    float kr;
    fresnel(dir, N, object->ior, kr);
    hitColor = reflectionColor * kr + refractionColor * (1 - kr);
    return hitColor;
}

Vec3f Trace::reflection(const Vec3f &dir, uint32_t &index, Vec2f &uv, Vec2f &st,
                        const Shape *hitObject, const Vec3f &hitPoint,
                        const Vec3f &N, const int depth) {
    Vec3f hitColor = g_options.backgroundColor;
    float kr = 0.5f;
    // fresnel(dir, N, hitObject->object.ior, kr);
    Vec3f reflectionDirection = dir.reflect(N);
    Vec3f reflectionRayOrig = (reflectionDirection.dot(N) < 0)
                                  ? hitPoint + N * g_options.bias
                                  : hitPoint - N * g_options.bias;
    hitColor = castRay(reflectionRayOrig.normalized(),
                       reflectionDirection.normalized(), depth + 1) *
               kr;
    return hitColor;
}

Vec3f Trace::diffuseAndGlossy(const Vec3f &dir, uint32_t &index, Vec2f &uv,
                              Vec2f &st, const Shape *hitObject,
                              const Vec3f &hitPoint, const Vec3f &N,
                              const int depth) {
    const Object *object = hitObject->getObject();

    Vec3f hitColor = g_options.backgroundColor;
    // [comment]
    // We use the Phong illumation model int the default case. The
    // phong model is composed of a diffuse and a specular
    // reflection component.
    // [/comment]
    Vec3f lightAmt = Vec3f(0.0f, 0.0f, 0.0f);
    Vec3f specularColor = Vec3f(0.f, 0.0f, 0.0f);
    Vec3f shadowPointOrig =
        (dir.dot(N) < 0) ? N * g_options.bias : hitPoint - N * g_options.bias;
    // [comment]
    // Loop over all lights in the scene and sum their contribution
    // up We also apply the lambert cosine law here though we
    // haven't explained yet what this means.
    // [/comment]
    for (uint32_t i = 0; i < g_scene.lightCnt; ++i) {
        Vec3f lightDir = g_scene.lights[i].position - hitPoint;
        // square of the distance between hitPoint and the light
        float lightDistance2 = lightDir.dot(lightDir);
        lightDir = lightDir.normalized();
        float LdotN = fmaxf(0.f, lightDir.dot(N));
        Shape *shadowHitObject = nullptr;
        float tNearShadow = INF;
        // is the point in shadow, and is the nearest occluding
        // object closer to the object than the light itself?
        bool inShadow = trace(shadowPointOrig, lightDir, tNearShadow, index, uv,
                              &shadowHitObject) &&
                        tNearShadow * tNearShadow < lightDistance2;
        lightAmt += g_scene.lights[i].intensity * LdotN * (1 - inShadow);
        Vec3f reflectionDirection = (-lightDir).reflect(N);
        float dotp = fmaxf(0.f, -reflectionDirection.dot(dir));

        specularColor +=
            powf(dotp, object->specularExponent) * g_scene.lights[i].intensity;
    }

    Vec3f diffuse = hitObject->evalDiffuseColor(st);

    hitColor = lightAmt * diffuse * object->Kd +
               specularColor * object->Ks;
    return hitColor;
}

Vec3f Trace::castRay(const Vec3f &orig, const Vec3f &dir, uint32_t depth) {
    if (depth > g_options.maxDepth) {
        return g_options.backgroundColor;
    }
    Vec3f hitColor = g_options.backgroundColor;
    float tnear = INF;
    Vec2f uv;
    uint32_t index = 0;
    Shape *hitObject = nullptr;
    if (trace(orig, dir, tnear, index, uv, &hitObject)) {
        Vec3f hitPoint = orig + dir * tnear;
        Vec3f N;   // normal
        Vec2f st;  // st coordinates

        // hitObject->getSurfaceProperties(hitPoint, dir, index, uv, N, st);
        hitObject->getSurfaceProperties(hitPoint, dir, index, uv, N, st);

        Object::MaterialType material = hitObject->getObject()->materialType;
        switch (material) {
            case Object::REFLECTION_AND_REFRACTION:
                return reflectionAndRefraction(dir, index, uv, st, hitObject,
                                               hitPoint, N, depth);
            case Object::REFLECTION:
                return reflection(dir, index, uv, st, hitObject, hitPoint, N,
                                  depth);
            case Object::DIFFUSE_AND_GLOSSY:
                return diffuseAndGlossy(dir, index, uv, st, hitObject, hitPoint,
                                        N, depth);
            default:
                return Vec3f(1.0f, 0.5f, 0.25f);
        }
    }

    return hitColor;
}

__device__ float randk(hiprandState *const localState) {
    return hiprand(localState) / INT32_MAX;
    // return 0.5f;
}

__global__ void kernel(unsigned char *surface, hiprandState *const rngStates) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if (x >= g_options.width || y >= g_options.height) return;

    // get a pointer to the pixel at (x,y)
    float *pixel = (float *)(surface + y * C_PITCH) + 4 * x;
    pixel[RED] = 0.0f;
    pixel[GREEN] = 0.0f;
    pixel[BLUE] = 0.0f;
    pixel[ALPHA] = 1.0f;

    float widthScale = 1 / (float)g_options.width;
    float heightScale = 1 / (float)g_options.height;

    hiprandState *localState =
        rngStates + threadIdx.y * blockDim.x + threadIdx.x;
    float ndcX = (2.0f * (x + 0.5f) * widthScale - 1.0f) *
                 g_options.imageAspectRatio * g_options.scale;
    float ndcY = (1.0f - 2.0f * (y + 0.5f) * heightScale) * g_options.scale;

    for (int i = 0; i < g_options.samples; ++i) {
        float xJitter = g_options.samples > 1 ? randk(localState) - 0.5 : 0.0f;
        float yJitter = g_options.samples > 1 ? randk(localState) - 0.5 : 0.0f;

        Vec3f dir =
            Vec3f(ndcX + xJitter * widthScale, ndcY + yJitter * heightScale, 1)
                .normalized();
        dir = g_scene.camera.multVec(dir);
        dir = dir.normalized();
        Trace trace(surface);
        Vec3f result = trace.castRay(g_scene.orig, dir, 0);

        pixel[RED] += result.data[Vec3f::X];
        pixel[GREEN] += result.data[Vec3f::Y];
        pixel[BLUE] += result.data[Vec3f::Z];
    }

    for (int i = 0; i < ALPHA; ++i) {
        pixel[i] /= g_options.samples;
    }
}

void cudamain(const Options &options, const Scene &scene, const void *surface,
              size_t pitch, int blockDim, unsigned char *rngStates) {
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(C_PITCH), &pitch, sizeof(size_t)));

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_options), &options, sizeof(Options)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(g_scene), &scene, sizeof(Scene)));
    gpuErrchk(hipPeekAtLastError());

    dim3 threads = dim3(blockDim, blockDim);
    dim3 grids = dim3((options.width + threads.x - 1) / threads.x,
                      (options.height + threads.y - 1) / threads.y);

    // fprintf(stderr, "width: %d, height: %d, threads: %d, %d grids: %d, %d\n",
    // width, height, threads.x, threads.y,
    //        grids.x, grids.y);
    kernel<<<grids, threads>>>((unsigned char *)surface,
                               (hiprandState *)rngStates);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

// RNG init kernel
__global__ void cuke_initRNG(hiprandState *const rngStates,
                             const unsigned int seed, int blkXIdx) {
    // Determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int x = blkXIdx * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    tid = x * gridDim.x + y;

    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}

unsigned char *cu_initCurand(int width, int height) {
    hipError_t error = hipSuccess;

    dim3 block = dim3(16, 16);  // block dimensions are fixed to be 256 threads
    dim3 grid =
        dim3((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // init hiprand
    hiprandState *rngStates = NULL;
    hipError_t cudaResult = hipMalloc(
        (void **)&rngStates,
        grid.x * block.x * /*grid.y * block.y **/ sizeof(hiprandState));
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    unsigned int seed = 1234;

    for (int blkXIdx = 0; blkXIdx < grid.x; blkXIdx++) {
        cuke_initRNG<<<dim3(1, grid.y), block>>>(rngStates, seed, blkXIdx);
    }

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return (unsigned char *)rngStates;
}

void cu_cleanCurand(unsigned char *p_rngStates) {
    // cleanup
    if (p_rngStates) {
        hipFree(p_rngStates);
    }
}